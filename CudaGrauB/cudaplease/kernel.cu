
#include "hip/hip_runtime.h"


#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <list>
#include <stdio.h>
using namespace std;

int* sample;
int sampleW;
int sampleH;
int* sampleR;
int* sampleG;
int* sampleB;

int* target;
int targetW;
int targetH;
int* targetR;
int* targetG;
int* targetB;

int* resultR;
int* resultG;
int* resultB;

int sizeAdj = 1;

__global__ void addKernel(int* imgW, int* imgH, float* distances, int* sampleR, int* sampleG, int* sampleB, int* targetR, int* targetG, int* targetB, int* adjSize)
{
	printf("Entered the Kernel");
	float dist = 0;
	int adj = 0;
	for (int i = -*adjSize; i <= *adjSize; i++)
	{
		for (int j = -*adjSize; j <= *adjSize; j++, adj++)
		{
			if (i != 0 && j != 0)
			{
				int aX = *imgW + j;
				int aY = *imgH + i;

				if (aX < 0)
				{
					aX = *imgW - 1;
				}
				else if (aX >= *imgW)
				{
					aX = 0;
				}

				if (aY == -1)
				{
					aY = *imgH - 1;
				}
				else if (aY == *imgH)
				{
					aY = 0;
				}
				int r = sampleR[aY * blockDim.x + aX] - targetR[adj];
				int g = sampleG[aY * blockDim.x + aX] - targetG[adj];
				int b = sampleB[aY * blockDim.x + aX] - targetB[adj];
				dist += sqrtf(r*r + g*g + b*b);

			}
		}
	}

	distances[blockIdx.x * blockDim.x + threadIdx.x] = dist;
}


vector<int> GetPixelsSample(int sizeAdj, int sizeW, int sizeH)
{
	vector<int> pixelsArr;
	for (int i = -sizeAdj; i <= sizeAdj; i++)
	{
		for (int j = -sizeAdj; j <= sizeAdj; j++)
		{
			if (i != 0 && j != 0)
			{
				int aX = sizeW + j;
				int aY = sizeH + i;

				if (aX < 0)
				{
					aX = sampleW - 1;
				}
				else if (aX >= sampleW)
				{
					aX = 0;
				}

				if (aY == -1)
				{
					aY = sampleH - 1;
				}
				else if (aY == sampleH)
				{
					aY = 0;
				}
				pixelsArr.push_back(sample[aY * sampleW + aX]);
			}
		}
	}
	
	return pixelsArr;
}

vector<int> GetPixelsTarget(int sizeAdj, int sizeW, int sizeH)
{
	vector<int> pixelsArr;
	for (int i = -sizeAdj; i <= sizeAdj; i++)
	{
		for (int j = -sizeAdj; j <= sizeAdj; j++)
		{
			if (i != 0 && j != 0)
			{
				int aX = sizeW + j;
				int aY = sizeH + i;

				if (aX < 0)
				{
					aX = targetW - 1;
				}
				else if (aX >= targetW)
				{
					aX = 0;
				}

				if (aY == -1)
				{
					aY = targetH - 1;
				}
				else if (aY == targetH)
				{
					aY = 0;
				}
				pixelsArr.push_back(target[aY * targetW + aX]);
			}
		}
	}

	return pixelsArr;
}

void ReadFile(string sample_, string target_)
{

	ifstream file1, file2;

	string s, t;

	file1.open(sample_);
	cout << "==Reading the sample file (1/6) - Process 1 out of xx ==" << endl;
	if (!file1)
	{
		return;
	}
	cout << "==Read the sample file (2/6) - Process 1 out of x ==" << endl;

	file1 >> s;
	char buffer[1000];
	bool invalid = true;
	do
	{
		file1 >> s;
		if (s == "#")
		{
			file1.getline(buffer, 100);
		}
		else
		{
			invalid = false;
		}
	} while (invalid);

	sampleW = atoi(s.c_str());
	file1 >> sampleH >> s;

	int sampleArrSize = sampleW * sampleH;
	sample = new int[sampleArrSize];
	sampleR = new int[sampleArrSize];
	sampleG = new int[sampleArrSize];
	sampleB = new int[sampleArrSize];

	for (int j = 0; j < sampleH; j++)
	{
		for (int i = 0; i < sampleW; i++)
		{
			file1 >> s;
			sampleR[j * sampleW + i] = stoi(s);
			file1 >> s;
			sampleG[j * sampleW + i] = stoi(s);
			file1 >> s;
			sampleB[j * sampleW + i] = stoi(s);
		}
	}
	file1.close();

	cout << "==Finished with the sample file (3/6) - Process 1 out of x ==" << endl;

	file2.open(target_);
	cout << "==Reading the texture file (4/6) - Process 1 out of x ==" << endl;
	if (!file2)
	{
		cout << "==ERROR: COULDNT READ TEXTURE==" << endl;
		return;
	}
	cout << "==Read the target file (5/6) - Process 1 out of x ==" << endl;

	file2 >> t;

	invalid = true;
	do
	{
		file2 >> t;
		if (t == "#")
		{
			file2.getline(buffer, 100);
		}
		else
		{
			invalid = false;
		}
	} while (invalid);


	targetW = atoi(t.c_str());
	file2 >> targetH >> t;

	int targetArrSize = targetW * targetH;
	target = new int[targetArrSize];
	targetR = new int[targetArrSize];
	targetG = new int[targetArrSize];
	targetB = new int[targetArrSize];

	for (int j = 0; j < targetH; j++)
	{
		for (int i = 0; i < targetW; i++)
		{
			file2 >> t;
			targetR[j * targetW + i] = stoi(t);
			file2 >> t;
			targetG[j * targetW + i] = stoi(t);
			file2 >> t;
			targetB[j * targetW + i] = stoi(t);
		}
	}
	file2.close();

	cout << "==Finished with the target file (6/6) - Process 1 out of sdwqedq ==" << endl;
	return;
}

void WriteFile(string filename)
{
	cout << "Entered the file writing function!" << endl;
	ofstream resultfile;

	resultfile.open(filename);

	resultfile << "P3" << endl << "# Criado para cadeira de Arquiteturas Gr�ficas, por Jo�o Rothmann" << endl << targetW << " " << targetH << endl << "255" << endl;

	for (int j = 0; j < targetH; j++)
	{
		for (int i = 0; i < targetW; i++)
		{
			int pos = j * targetW + i;
			resultfile << resultR[pos] << " " << resultG[pos] << " " << resultB[pos] << " " << endl;
		}
	}

	cout << "targetW: " << targetW << endl;
	cout << "targetH: " << targetH << endl;
	cout << "sampleW: " << targetW << endl;
	cout << "sampleH: " << targetH << endl;
	resultfile.close();
}

float CalculateDistance(int tr, int tg, int tb, int sr, int sg, int sb)
{
	float distance = float(((tr - sr) * (tr - sr)) + ((tg - sg) * (tg - sg)) + ((tb - sb) * (tb - sb)));
	return powf(distance, 0.5f);
}




// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int* imgW_, int* imgH_, int* targetW_, int* targetH_, float* distances_, int* sampleR_, int* sampleG_, int* sampleB_, int* targetR_, int* targetG_, int* targetB_, int* adjSize_)
{
	printf("Entered Cuda\n");
	int* imgW = 0;
	int* imgH = 0;
	int* targetW = 0;
	int* targetH = 0;
	float* distances = 0;
	int* sampleR = 0;
	int* sampleG = 0;
	int* sampleB = 0;
	int* targetR = 0;
	int* targetG = 0;
	int* targetB = 0;
	int* adjSize = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&imgW, sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&imgH, sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	cudaStatus = hipMalloc((void**)&targetW, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&targetH, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&distances, *imgW_ * *imgH_ * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&sampleR, *imgW_ * *imgH_ * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&sampleG, *imgW_ * *imgH_ * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&sampleB, *imgW_ * *imgH_ * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&targetR, *targetW_ * *targetH_ * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&targetG, *targetW_ * *targetH_ * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&targetB,  *targetW_ * *targetH_ * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&adjSize, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

    // Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(imgW, imgW_, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(imgH, imgH_, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}   
	cudaStatus = hipMemcpy(targetW, targetW_, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(targetH, targetH_, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}   
	cudaStatus = hipMemcpy(distances, distances_, *imgW_ * *imgH_ * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(sampleR, sampleR_, *imgW_ * *imgH_ * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}  
	cudaStatus = hipMemcpy(sampleG, sampleG_, *imgW_ * *imgH_ * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(sampleB, sampleB_, *imgW_ * *imgH_ * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}   
	
	cudaStatus = hipMemcpy(targetR, targetR_, *targetW_ * *targetH_ * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(targetG, targetG_, *targetW_ * *targetH_ * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}  
	cudaStatus = hipMemcpy(targetB, targetB_, *targetW_ * *targetH_ * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(adjSize, adjSize_, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<*imgW_, *imgH_>>>(imgW,imgH,distances,sampleR,sampleG,sampleB,targetR,targetG,targetB,adjSize);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(distances_, distances, *imgW * *imgH * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
	hipFree(imgW);
	hipFree(imgH);
	hipFree(targetW);
	hipFree(targetH);
	hipFree(distances);
	hipFree(sampleR);
	hipFree(sampleG);
	hipFree(sampleB);
	hipFree(targetR);
	hipFree(targetG);
	hipFree(targetB);
	hipFree(adjSize);
    
    return cudaStatus;
}

void CreateTexture()
{
	cout << "Entered the texture creation function2222" << endl;
	resultR = new int[targetW*targetH];
	resultG = new int[targetW*targetH];
	resultB = new int[targetW*targetH];

	cout << "==Declared result's RGB vectors (1/x) - Process 2 out of x ==" << endl;
	cout << "targetW: " << targetW << endl;
	cout << "targetH: " << targetH << endl;
	cout << "----" << endl;
	cout << "sampleW: " << sampleW << endl;
	cout << "sampleH: " << sampleH << endl;


	float* difference = new float[sampleW*sampleH];

	for (int j = 0; j < targetH; j++)
	{
		for (int i = 0; i < targetW; i++)
		{
			int* bestPixelPos = 0;
			vector<int> pixelsTarget = GetPixelsTarget(1, targetW, targetH);
			float lowestDistance = INFINITY;

			hipError_t cudaStatus = addWithCuda(&sampleW, &sampleH, &targetW, &targetH, difference, sampleR, sampleG, sampleB, targetR, targetG, targetB, &sizeAdj);
			if (cudaStatus != hipSuccess)
			{
				return;
			}

			for (int y = 0; y < sampleH; y++)
			{
				for (int x = 0; x < sampleW; x++)
				{
					float diff = difference[y * sampleW + x];
					if (diff < lowestDistance)
					{
						lowestDistance = diff;
						bestPixelPos = &sample[y * sampleW + x];
					}
				}
			}

			resultR[j * sampleW + i] = sampleR[*bestPixelPos];
			resultG[j * sampleW + i] = sampleG[*bestPixelPos];
			resultB[j * sampleW + i] = sampleB[*bestPixelPos];

			//cout << "Pixel color: " << j * sampleW + i << " - " << resultR[j * sampleW + i] << " " << resultG[j * sampleW + i] << " " << resultB[j * sampleW + i] << endl;
		}
	}
	return;
}


int main()
{
	ReadFile("sample.ppm", "texture.ppm");
	CreateTexture();
	WriteFile("output.ppm");
	cout << "End of the program" << endl;
	system("pause");
	return 0;
}